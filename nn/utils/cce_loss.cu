#include "hip/hip_runtime.h"
#include "cce_loss.cuh"
#include "exception.cuh"

#include <math.h>
#include <iostream>
#include <assert.h>

__global__ void dCategoricalCrossEntropyCost(float *predictions, float *target, float *dY,
											 int size){

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		dY[index] = (predictions[index] - target[index]);
	}
}

__global__ void categoricalCrossEntropyCost(float *predictions, float *target,
											int size, float *cost){

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
	{
		float partial_cost = target[index] * logf(predictions[index]);
		atomicAdd(cost, -partial_cost / size);
	}
}

float CCELoss::cost(Tensor predictions, Tensor target){
	assert(predictions.shape.x == target.shape.x && predictions.shape.y == target.shape.y);

	float *cost;
	hipMallocManaged(&cost, sizeof(float));
	*cost = 0.0f;

	dim3 block_size(256);
	dim3 num_of_blocks((predictions.shape.x * predictions.shape.y + block_size.x - 1) / block_size.x);
	categoricalCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device.get(),
															   target.data_device.get(),
															   predictions.shape.x * predictions.shape.y, cost);
	hipDeviceSynchronize();
	NNException::throwIfDeviceErrorsOccurred("Cannot compute categorical cross entropy cost.");

	float cost_value = *cost;
	hipFree(cost);

	return cost_value;
}

Tensor CCELoss::dCost(Tensor predictions, Tensor target, Tensor dY){
	assert(predictions.shape.x == target.shape.x && predictions.shape.y == target.shape.y);

	dim3 block_size(256);
	dim3 num_of_blocks((predictions.shape.x * predictions.shape.y + block_size.x - 1) / block_size.x);
	dCategoricalCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device.get(),
																target.data_device.get(),
																dY.data_device.get(),
																predictions.shape.x * predictions.shape.y);
	NNException::throwIfDeviceErrorsOccurred("Cannot compute derivative for categorical cross entropy.");
	return dY;
}
