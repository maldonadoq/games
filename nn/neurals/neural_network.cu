#include "neural_network.cuh"

NeuralNetwork::NeuralNetwork(float learning_rate) :
	learning_rate(learning_rate)
{ }

NeuralNetwork::~NeuralNetwork() {
	for (auto layer : layers) {
		delete layer;
	}
}

void NeuralNetwork::addLayer(NNLayer* layer) {
	this->layers.push_back(layer);
}

Tensor NeuralNetwork::forward(Tensor X) {
	Tensor Z = X;

	for (auto layer : layers) {
		Z = layer->forward(Z);
	}

	Y = Z;
	return Y;
}

void NeuralNetwork::backprop(Tensor predictions, Tensor target) {
	dY.allocateMemoryIfNotAllocated(predictions.shape);

	Tensor error = cce_cost.dCost(predictions, target, dY);

	for (auto it = this->layers.rbegin(); it != this->layers.rend(); it++) {
		error = (*it)->backprop(error, learning_rate);
	}

	hipDeviceSynchronize();
}

std::vector<NNLayer*> NeuralNetwork::getLayers() const {
	return layers;
}
